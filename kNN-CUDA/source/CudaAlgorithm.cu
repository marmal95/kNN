#include "hip/hip_runtime.h"
#include "CudaAlgorithm.hpp"
#include "hip/hip_runtime.h"
#include ""
#include <cfloat>

__device__ float distanceBetweenPointsOnCUDA(
	const float* trainingData, float* testingData,
	const std::size_t trainingDataIndex, const std::size_t testingDataIndex,
	const std::size_t rowSize)
{
	float sum = 0.f;
	for (int i = 0; i < rowSize - 2; ++i)
	{
		sum += (trainingData[trainingDataIndex * rowSize + i] - testingData[testingDataIndex * rowSize + i])
			* (trainingData[trainingDataIndex * rowSize + i] - testingData[testingDataIndex * rowSize + i]);
	}

	return sqrtf(sum);
}

__global__ void minMaxNormalizationOnCUDA(float* data, const float* minMax, const std::size_t rowSize, const std::size_t dataSize)
{
	const auto rowIdx = threadIdx.x + blockIdx.x * blockDim.x;

	if (rowIdx < dataSize)
	{
		for (auto featureIdx = 0u; featureIdx < rowSize - 2; ++featureIdx)
		{
			data[rowIdx * rowSize + featureIdx] = (data[rowIdx * rowSize + featureIdx] - minMax[2 * featureIdx])
				/ (minMax[2 * featureIdx + 1] - minMax[2 * featureIdx]);
		}
	}
}

__global__ void findMinMaxOnCuda(float* rows, float* minMaxs, const std::size_t rowSize, const std::size_t dataSize)
{
	const auto featureId = threadIdx.x + blockIdx.x * blockDim.x;
	const auto numOfFeatures = rowSize - 2;

	if (featureId < numOfFeatures)
	{
		auto min = FLT_MAX;
		auto max = FLT_MIN;

		for (auto rowIndex = 0u; rowIndex < dataSize; rowIndex++)
		{
			const auto value = rows[rowIndex * rowSize + featureId];
			if (value < min)
			{
				min = value;
			}
			if (value > max)
			{
				max = value;
			}
			minMaxs[featureId * 2] = min;
			minMaxs[featureId * 2 + 1] = max;
		}
	}
}

__global__ void knnOnCuda(
	const float* trainingData, float* testingData,
	const std::size_t trainingDataSize, const std::size_t testingDataSize,
	const std::size_t rowSize)
{
	const auto testRowIdx = threadIdx.x + blockIdx.x * blockDim.x;
	auto smallestDistance = DBL_MAX;
	auto nearestLabel = 0u;

	if (testRowIdx < testingDataSize)
	{
		for (auto trainIndex = 0u; trainIndex < trainingDataSize; ++trainIndex)
		{
			auto distance = distanceBetweenPointsOnCUDA(trainingData, testingData, trainIndex, testRowIdx, rowSize);
			if (distance < smallestDistance)
			{
				smallestDistance = distance;
				nearestLabel = trainingData[trainIndex * rowSize + rowSize - 2];
			}
		}

		testingData[testRowIdx * rowSize + rowSize - 1] = nearestLabel;
	}
}

void Cuda::knn(const FlatDataView& trainingData, FlatDataView& testingData)
{
	constexpr int NUM_OF_THREADS = 1024;
	const auto BLOCK_SIZE = std::ceil(testingData.getNumberOfRows() / NUM_OF_THREADS) + 1;

	const auto& trainingRawData = *trainingData;
	auto& testingRawData = *testingData;

	float* deviceTrainingData = nullptr;
	float* deviceTestingData = nullptr;

	hipMalloc((void**)&deviceTrainingData, trainingRawData.size() * sizeof(float));
	hipMalloc((void**)&deviceTestingData, testingRawData.size() * sizeof(float));

	hipMemcpy(deviceTrainingData, trainingRawData.data(), trainingRawData.size() * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceTestingData, testingRawData.data(), testingRawData.size() * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	knnOnCuda << < BLOCK_SIZE, NUM_OF_THREADS >> > (deviceTrainingData, deviceTestingData, trainingData.getNumberOfRows(), testingData.getNumberOfRows(), testingData.getRowSize());

	hipEventRecord(stop);
	hipDeviceSynchronize();
	hipEventSynchronize(stop);

	const auto hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
	{
		std::cout << "FAIL!! " << hipGetErrorString(hipError_t) << std::endl;
	}

	hipMemcpy(testingRawData.data(), deviceTestingData, testingRawData.size() * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(deviceTrainingData);
	hipFree(deviceTestingData);

	float timeMs{};
	hipEventElapsedTime(&timeMs, start, stop);
	std::cout << "[CUDA] (only CUDA calculations): " << timeMs << " ms" << std::endl;
}

void Cuda::minMax(FlatDataView& data)
{
	constexpr int NUM_OF_THREADS = 1024;
	const auto BLOCK_SIZE = std::ceil(data.getNumberOfRows() / NUM_OF_THREADS) + 1;
	auto& rawData = *data;

	float* dev_rows = nullptr;
	float* dev_minMax = nullptr;

	hipMalloc((void**)&dev_rows, rawData.size() * sizeof(float));
	hipMalloc((void**)&dev_minMax, (data.getRowSize() - 2) * 2 * sizeof(float));
	hipMemcpy(dev_rows, rawData.data(), rawData.size() * sizeof(float), hipMemcpyHostToDevice);

	findMinMaxOnCuda << < BLOCK_SIZE, NUM_OF_THREADS >> > (dev_rows, dev_minMax, data.getRowSize(), data.getNumberOfRows());
	minMaxNormalizationOnCUDA << < BLOCK_SIZE, NUM_OF_THREADS >> > (dev_rows, dev_minMax, data.getRowSize(), data.getNumberOfRows());

	auto hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
	{
		std::cout << "FAIL!! " << hipGetErrorString(hipError_t) << std::endl;
	}

	hipMemcpy(rawData.data(), dev_rows, rawData.size() * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_rows);
	hipFree(dev_minMax);
}